#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>
#include <jni.h>
#include "org_dromara_jcudax_JCudax.h"


extern "C" {
    // ====================== matrixSoftMaxPd ======================
    __global__ void matrixSoftMaxPdKernel(double* qkt, double* errorMatrix, double* grMatrix, int x, int y, double param) {
        int i = blockIdx.x * blockDim.x + threadIdx.x; // 行索引
        int j = blockIdx.y * blockDim.y + threadIdx.y; // 列索引

        if (i < x && j < y) {
            int z = y;  // qr 的列数等于 y
            double jValue = qkt[i * y + j];
            double sigma = 0;

            for (int k = 0; k < z; ++k) {
                double kValue = qkt[i * y + k];
                double error = errorMatrix[i * y + k];
                double er;

                if (k != j) {
                    er = -error * kValue * jValue;
                } else {
                    er = jValue * (1.0 - jValue) * error;
                }

                sigma += er;
            }

            grMatrix[i * y + j] = sigma / param;
        }
    }

    void matrixSoftMaxPd(double* h_qkt, double* h_errorMatrix, double* h_grMatrix,  int x, int y, double wordVectorDimension) {
        double param = sqrt(wordVectorDimension);
        size_t size = x * y * sizeof(double);

        double *d_qkt, *d_errorMatrix, *d_grMatrix;

        // 分配 GPU 内存
        hipMalloc(&d_qkt, size);
        hipMalloc(&d_errorMatrix, size);
        hipMalloc(&d_grMatrix, size);

        // 复制数据到 GPU
        hipMemcpy(d_qkt, h_qkt, size, hipMemcpyHostToDevice);
        hipMemcpy(d_errorMatrix, h_errorMatrix, size, hipMemcpyHostToDevice);

        // 定义线程和块
        dim3 blockSize(16, 16);
        dim3 gridSize((x + blockSize.x - 1) / blockSize.x, (y + blockSize.y - 1) / blockSize.y);

        // 启动 CUDA 内核
        matrixSoftMaxPdKernel<<<gridSize, blockSize>>>(d_qkt, d_errorMatrix, d_grMatrix, x, y, param);

        // 复制结果回主机
        hipMemcpy(h_grMatrix, d_grMatrix, size, hipMemcpyDeviceToHost);

        // 释放 GPU 内存
        hipFree(d_qkt);
        hipFree(d_errorMatrix);
        hipFree(d_grMatrix);
    }



    JNIEXPORT void JNICALL Java_org_dromara_jcudax_JCudax_matrixSoftMaxPd  (
        JNIEnv* env, jclass cls,
        jdoubleArray j_qkt, jdoubleArray j_errorMatrix, jdoubleArray j_grMatrix,
        jint x, jint y, jdouble wordVectorDimension) {

        // 获取 Java 数组数据
        jdouble* h_qkt = env->GetDoubleArrayElements(j_qkt, NULL);
        jdouble* h_errorMatrix = env->GetDoubleArrayElements(j_errorMatrix, NULL);
        jdouble* h_grMatrix = env->GetDoubleArrayElements(j_grMatrix, NULL);

        // 调用 CUDA 实现
        matrixSoftMaxPd(h_qkt, h_errorMatrix, h_grMatrix, x, y, wordVectorDimension);

        // 将结果同步回 Java 数组
        env->ReleaseDoubleArrayElements(j_qkt, h_qkt, 0);
        env->ReleaseDoubleArrayElements(j_errorMatrix, h_errorMatrix, 0);
        env->ReleaseDoubleArrayElements(j_grMatrix, h_grMatrix, 0);
    }


    // ====================== matrixSoftMaxPdFp32 ======================
    __global__ void matrixSoftMaxPdKernelFp32(float* qkt, float* errorMatrix, float* grMatrix, int x, int y, float param) {
        int i = blockIdx.x * blockDim.x + threadIdx.x; // 行索引
        int j = blockIdx.y * blockDim.y + threadIdx.y; // 列索引

        if (i < x && j < y) {
            int z = y;  // qr 的列数等于 y
            float jValue = qkt[i * y + j];
            float sigma = 0;

            for (int k = 0; k < z; ++k) {
                float kValue = qkt[i * y + k];
                float error = errorMatrix[i * y + k];
                float er;

                if (k != j) {
                    er = -error * kValue * jValue;
                } else {
                    er = jValue * (1.0 - jValue) * error;
                }

                sigma += er;
            }

            grMatrix[i * y + j] = sigma / param;
        }
    }

    void matrixSoftMaxPdFp32(float* h_qkt, float* h_errorMatrix, float* h_grMatrix,  int x, int y, float wordVectorDimension) {
        float param = sqrt(wordVectorDimension);
        size_t size = x * y * sizeof(float);

        float *d_qkt, *d_errorMatrix, *d_grMatrix;

        // 分配 GPU 内存
        hipMalloc(&d_qkt, size);
        hipMalloc(&d_errorMatrix, size);
        hipMalloc(&d_grMatrix, size);

        // 复制数据到 GPU
        hipMemcpy(d_qkt, h_qkt, size, hipMemcpyHostToDevice);
        hipMemcpy(d_errorMatrix, h_errorMatrix, size, hipMemcpyHostToDevice);

        // 定义线程和块
        dim3 blockSize(16, 16);
        dim3 gridSize((x + blockSize.x - 1) / blockSize.x, (y + blockSize.y - 1) / blockSize.y);

        // 启动 CUDA 内核
        matrixSoftMaxPdKernelFp32<<<gridSize, blockSize>>>(d_qkt, d_errorMatrix, d_grMatrix, x, y, param);

        // 复制结果回主机
        hipMemcpy(h_grMatrix, d_grMatrix, size, hipMemcpyDeviceToHost);

        // 释放 GPU 内存
        hipFree(d_qkt);
        hipFree(d_errorMatrix);
        hipFree(d_grMatrix);
    }



    JNIEXPORT void JNICALL Java_org_dromara_jcudax_JCudax_matrixSoftMaxPdFp32  (
        JNIEnv* env, jclass cls,
        jfloatArray j_qkt, jfloatArray j_errorMatrix, jfloatArray j_grMatrix,
        jint x, jint y, jfloat wordVectorDimension) {

        // 获取 Java 数组数据
        jfloat* h_qkt = env->GetFloatArrayElements(j_qkt, NULL);
        jfloat* h_errorMatrix = env->GetFloatArrayElements(j_errorMatrix, NULL);
        jfloat* h_grMatrix = env->GetFloatArrayElements(j_grMatrix, NULL);

        // 调用 CUDA 实现
        matrixSoftMaxPdFp32(h_qkt, h_errorMatrix, h_grMatrix, x, y, wordVectorDimension);

        // 将结果同步回 Java 数组
        env->ReleaseFloatArrayElements(j_qkt, h_qkt, 0);
        env->ReleaseFloatArrayElements(j_errorMatrix, h_errorMatrix, 0);
        env->ReleaseFloatArrayElements(j_grMatrix, h_grMatrix, 0);
    }
}
